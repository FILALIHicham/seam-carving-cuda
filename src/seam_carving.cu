#include "hip/hip_runtime.h"
#include "seam_carving.h"
#include "gpu_memory.h"
#include "energy.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

// Kernel to compute cumulative energy map
__global__ void compute_row_cumulative_energy(
    const float *energy,
    float *cumulative_energy,
    int *backtrack,
    int width,
    int height,
    int current_row,
    int direction
) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    if (x >= width) return;

    int idx, energy_idx;
    if (direction == 0) {  // Vertical seam
        idx = current_row * width + x;
        energy_idx = idx;
    } else {  // Horizontal seam
        idx = current_row + x * height;  // Transposed layout
        energy_idx = current_row * width + x;  
    }

    if (current_row == 0) {
        // First row just copies the energy values
        cumulative_energy[idx] = energy[energy_idx];
        backtrack[idx] = x;
        return;
    }

    // Neighbor calculations
    float left, middle, right;
    if (direction == 0) {  // Vertical seam
        int prev_row = (current_row - 1) * width;
        left = (x > 0) ? cumulative_energy[prev_row + (x - 1)] : INFINITY;
        middle = cumulative_energy[prev_row + x];
        right = (x < width - 1) ? cumulative_energy[prev_row + (x + 1)] : INFINITY;
    } else {  // Horizontal seam
        int prev_row = current_row - 1;
        left = (x > 0) ? cumulative_energy[prev_row + (x - 1) * height] : INFINITY;
        middle = cumulative_energy[prev_row + x * height];
        right = (x < width - 1) ? cumulative_energy[prev_row + (x + 1) * height] : INFINITY;
    }

    // Find minimum of the three possible paths
    float min_energy = middle;
    int min_x = x;

    if (left < min_energy) {
        min_energy = left;
        min_x = x - 1;
    }
    if (right < min_energy) {
        min_energy = right;
        min_x = x + 1;
    }

    // Store cumulative energy and backtrack pointer
    cumulative_energy[idx] = energy[energy_idx] + min_energy;
    backtrack[idx] = min_x;
}

int* remove_seam_with_path(Image *img, float *device_energy, int direction) {
    int width = img->width;
    int height = img->height;
    int seam_length = direction == 0 ? height : width;
    int search_width = direction == 0 ? width : height;

    size_t cumulative_size = width * height * sizeof(float);
    size_t backtrack_size = width * height * sizeof(int);

    // Allocate GPU memory
    float *cumulative_energy;
    int *backtrack;
    allocate_device_memory((void **)&cumulative_energy, cumulative_size);
    allocate_device_memory((void **)&backtrack, backtrack_size);

    // Compute cumulative energy row by row
    dim3 block_dim(256, 1);
    dim3 grid_dim((search_width + block_dim.x - 1) / block_dim.x, 1);

    for (int row = 0; row < seam_length; row++) {
        compute_row_cumulative_energy<<<grid_dim, block_dim>>>(
            device_energy,
            cumulative_energy,
            backtrack,
            search_width,
            seam_length,
            row,
            direction
        );
        
        hipError_t err = hipGetLastError();
        if (err != hipSuccess) {
            fprintf(stderr, "Kernel error at row %d: %s\n", row, hipGetErrorString(err));
            exit(1);
        }
        
        hipDeviceSynchronize();
    }

    // Copy results back to host
    float *host_cumulative = (float *)malloc(cumulative_size);
    int *host_backtrack = (int *)malloc(backtrack_size);
    if (!host_cumulative || !host_backtrack) {
        fprintf(stderr, "Failed to allocate host memory\n");
        exit(1);
    }
    
    copy_to_host(host_cumulative, cumulative_energy, cumulative_size);
    copy_to_host(host_backtrack, backtrack, backtrack_size);

    // Find minimum energy in last row/column
    float min_energy = INFINITY;
    int seam_end = 0;
    int last_row_offset = (seam_length - 1) * (direction == 0 ? search_width : 1);

    for (int x = 0; x < search_width; x++) {
        float current = direction == 0 ?
            host_cumulative[last_row_offset + x] :
            host_cumulative[x * seam_length + (seam_length - 1)];
        
        if (current < min_energy) {
            min_energy = current;
            seam_end = x;
        }
    }

    // Backtrack to find seam path
    int *seam = (int *)malloc(seam_length * sizeof(int));
    if (!seam) {
        fprintf(stderr, "Failed to allocate seam path memory\n");
        exit(1);
    }
    
    seam[seam_length-1] = seam_end;

    for (int i = seam_length - 2; i >= 0; i--) {
        int curr_x = seam[i + 1];
        if (direction == 0) {
            seam[i] = host_backtrack[i * search_width + curr_x];
        } else {
            seam[i] = host_backtrack[curr_x * seam_length + i];
        }
    }

    // Remove seam from image
    unsigned char *new_data = NULL;

    if (direction == 0) {  // Vertical seam -> new width = width - 1
        new_data = (unsigned char *)malloc((width - 1) * height * 3);
    } else {               // Horizontal seam -> new height = height - 1
        new_data = (unsigned char *)malloc(width * (height - 1) * 3);
    }

    if (direction == 0) {  // Vertical seam
        for (int y = 0; y < height; y++) {
            int offset = 0;
            for (int x = 0; x < width; x++) {
                if (x == seam[y]) {
                    offset = -1;
                    continue;
                }
                for (int c = 0; c < 3; c++) {
                    new_data[(y * (width - 1) + (x + offset)) * 3 + c] = 
                        img->data[(y * width + x) * 3 + c];
                }
            }
        }
        img->width--;
    } else {  // Horizontal seam
        for (int y = 0; y < height; y++) {
            if (y == seam[0]) continue;
            
            int new_y = y > seam[0] ? y - 1 : y;
            for (int x = 0; x < width; x++) {
                for (int c = 0; c < 3; c++) {
                    new_data[(new_y * width + x) * 3 + c] = 
                        img->data[(y * width + x) * 3 + c];
                }
            }
        }
        img->height--;
    }

    // Update image
    free(img->data);
    img->data = new_data;

    // Cleanup
    free(host_cumulative);
    free(host_backtrack);
    free_device_memory(cumulative_energy);
    free_device_memory(backtrack);

    return seam;
}

// Function to store k seams before insertion
int** find_k_seams(Image *img, float *device_energy, int k, int direction, int *seam_lengths) {
    int width = img->width;
    int height = img->height;
    
    // Allocate array to store k seams
    int **seams = (int **)malloc(k * sizeof(int *));
    if (!seams) {
        fprintf(stderr, "Failed to allocate seams array\n");
        exit(1);
    }
    
    // Create a temporary image for seam calculation
    Image temp_img = {width, height, NULL};
    temp_img.data = (unsigned char *)malloc(width * height * 3);
    if (!temp_img.data) {
        fprintf(stderr, "Failed to allocate temporary image data\n");
        exit(1);
    }
    memcpy(temp_img.data, img->data, width * height * 3);
    
    // Calculate initial energy size
    size_t energy_size = width * height * sizeof(float);
    
    // Create temporary energy map
    float *temp_energy;
    allocate_device_memory((void **)&temp_energy, energy_size);
    
    // Copy initial energy map
    copy_to_device(device_energy, temp_energy, energy_size);
    
    // Find k seams
    for (int i = 0; i < k; i++) {
        // Compute energy for current state
        compute_energy(&temp_img, temp_energy);
        
        // Find seam
        seam_lengths[i] = direction == 0 ? height : width;
        seams[i] = remove_seam_with_path(&temp_img, temp_energy, direction);
        
        if (!seams[i]) {
            fprintf(stderr, "Failed to find seam %d\n", i);
            exit(1);
        }
        
        // Update energy map size after removal
        if (direction == 0) {
            width--;
            energy_size = width * height * sizeof(float);
        } else {
            height--;
            energy_size = width * height * sizeof(float);
        }
        
        // Reallocate temp_energy with new size
        float *new_temp_energy;
        allocate_device_memory((void **)&new_temp_energy, energy_size);
        free_device_memory(temp_energy);
        temp_energy = new_temp_energy;
        
        // Skip energy update for the last iteration
        if (i < k - 1) {
            compute_energy(&temp_img, temp_energy);
        }
    }
    
    // Cleanup temporary resources
    free_image(temp_img);
    free_device_memory(temp_energy);
    
    return seams;
}

__global__ void insert_seam_kernel(
    const unsigned char *input,
    unsigned char *output,
    const int *seam,
    int width,
    int height,
    int direction  // 0 for vertical, 1 for horizontal
) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    int max_idx = direction == 0 ? height : width;
    if (idx >= max_idx) return;
    
    if (direction == 0) {  // Vertical seam
        int y = idx;
        int seam_x = seam[y];
        
        // Process each pixel in the row
        for (int x = 0; x <= width; x++) {
            if (x == seam_x) {
                // Interpolate new pixel
                int left_x = max(0, x - 1);
                int right_x = min(width - 1, x);
                
                for (int c = 0; c < 3; c++) {
                    int left_val = input[(y * width + left_x) * 3 + c];
                    int right_val = input[(y * width + right_x) * 3 + c];
                    output[(y * (width + 1) + x) * 3 + c] = (left_val + right_val) / 2;
                }
            } else if (x < seam_x) {
                // Copy pixels before seam
                for (int c = 0; c < 3; c++) {
                    output[(y * (width + 1) + x) * 3 + c] = input[(y * width + x) * 3 + c];
                }
            } else { // x > seam_x
                // Copy pixels after seam
                for (int c = 0; c < 3; c++) {
                    output[(y * (width + 1) + x) * 3 + c] = input[(y * width + (x - 1)) * 3 + c];
                }
            }
        }
    } else {  // Horizontal seam
        int x = idx;
        int seam_y = seam[x];
        
        // Process each pixel in the column
        for (int y = 0; y <= height; y++) {
            if (y == seam_y) {
                // Interpolate new pixel
                int above_y = max(0, y - 1);
                int below_y = min(height - 1, y);
                
                for (int c = 0; c < 3; c++) {
                    int above_val = input[(above_y * width + x) * 3 + c];
                    int below_val = input[(below_y * width + x) * 3 + c];
                    output[(y * width + x) * 3 + c] = (above_val + below_val) / 2;
                }
            } else if (y < seam_y) {
                // Copy pixels above seam
                for (int c = 0; c < 3; c++) {
                    output[(y * width + x) * 3 + c] = input[(y * width + x) * 3 + c];
                }
            } else { // y > seam_y
                // Copy pixels below seam
                for (int c = 0; c < 3; c++) {
                    output[(y * width + x) * 3 + c] = input[((y - 1) * width + x) * 3 + c];
                }
            }
        }
    }
}

void insert_seam(Image *img, int *seam, int direction) {
    int width = img->width;
    int height = img->height;
    
    // Calculate sizes
    size_t input_size = width * height * 3;
    size_t output_size = direction == 0 ? 
        (width + 1) * height * 3 : 
        width * (height + 1) * 3;
    size_t seam_size = direction == 0 ? height * sizeof(int) : width * sizeof(int);
    
    // Allocate device memory
    unsigned char *d_input, *d_output;
    int *d_seam;
    
    allocate_device_memory((void **)&d_input, input_size);
    allocate_device_memory((void **)&d_output, output_size);
    allocate_device_memory((void **)&d_seam, seam_size);
    
    // Copy data to device
    copy_to_device(d_input, img->data, input_size);
    copy_to_device(d_seam, seam, seam_size);
    
    // Launch kernel
    int max_threads = 256;
    int num_blocks = direction == 0 ? 
        (height + max_threads - 1) / max_threads : 
        (width + max_threads - 1) / max_threads;
    
    insert_seam_kernel<<<num_blocks, max_threads>>>(
        d_input,
        d_output,
        d_seam,
        width,
        height,
        direction
    );
    
    // Check for kernel errors
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "Kernel error: %s\n", hipGetErrorString(err));
        exit(1);
    }
    
    // Synchronize device
    hipDeviceSynchronize();
    
    // Allocate and copy back result
    unsigned char *new_data = (unsigned char *)malloc(output_size);
    if (!new_data) {
        fprintf(stderr, "Failed to allocate host memory for output\n");
        exit(1);
    }
    
    copy_to_host(new_data, d_output, output_size);
    
    // Update image
    free(img->data);
    img->data = new_data;
    
    // Update dimensions
    if (direction == 0) {
        img->width++;
    } else {
        img->height++;
    }
    
    // Cleanup device memory
    free_device_memory(d_input);
    free_device_memory(d_output);
    free_device_memory(d_seam);
}