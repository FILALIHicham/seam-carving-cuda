#include "gpu_memory.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

// Allocates memory on the GPU
void allocate_device_memory(void **dev_ptr, size_t size) {
    hipError_t err = hipMalloc(dev_ptr, size);
    if (err != hipSuccess) {
        fprintf(stderr, "Error: hipMalloc failed: %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE); // Exit if memory allocation fails
    }
}

// Frees memory on the GPU
void free_device_memory(void *dev_ptr) {
    hipError_t err = hipFree(dev_ptr);
    if (err != hipSuccess) {
        fprintf(stderr, "Error: hipFree failed: %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE); 
    }
}

// Copies data from host (CPU) to device (GPU)
void copy_to_device(void *dst, const void *src, size_t size) {
    hipError_t err = hipMemcpy(dst, src, size, hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        fprintf(stderr, "Error: hipMemcpy (HostToDevice) failed: %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

// Copies data from device (GPU) to host (CPU)
void copy_to_host(void *dst, const void *src, size_t size) {
    hipError_t err = hipMemcpy(dst, src, size, hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        fprintf(stderr, "Error: hipMemcpy (DeviceToHost) failed: %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE); 
    }
}
